#include "hip/hip_runtime.h"

#include "main.cuh"


int main(int argc, char **argv) {

	// Create default stream for the chain
	hipStream_t stream_dmrs;
	hipStreamCreate(&stream_default);
	hipStreamCreate(&stream_dmrs);

	//For timing purpose
	timerInit();
	startTimer();

	int N_bits, N_ri;
	const int Qm = 6;					// 64QAM Modulation
	const int N_l = 1;					// Number of Layers

	// Physical layer cell identity (we need for generation of random sequence)
	int N_id_cell = 2;						// assume enodeB scheduled cell 2 for the UE
	int M_pusch_rb = 100;					// number of resource blocks assigned to the UE
	int n_s = 0;							// assume UE send on time slot 0
	int n_RNTI = 10;						// radio network temporary identifier given to the UE by enodeB(assume 10)
	// (UNUSED) int N_subfr = 0;						// Subframe number within a radio frame
	BYTE* inputBits_h = readBits(argc, argv[1], &N_bits);			//Get input bits from the text file
	BYTE* riBits_h = readBits(argc, argv[2], &N_ri);					//Get RI bits from the text file

	//hipMalloc & hipMemcpy for inputBits & RI_Bits to Device
	Byte *inputBits_d = 0, *riBits_d = 0;

	hipMalloc((void **)&inputBits_d, sizeof(Byte)*N_bits);
	hipMalloc((void **)&riBits_d, sizeof(Byte)*N_ri);
	Byte* c_d = 0;
	hipMalloc((void **)&c_d, sizeof(Byte)*N_bits);
	
	// Copy data to the device using different stream 
	hipStream_t stream_mem;
	hipStreamCreate(&stream_mem);
	hipMemcpyAsync(inputBits_d, inputBits_h, sizeof(Byte)*N_bits, hipMemcpyHostToDevice, stream_mem);
	hipMemcpyAsync(riBits_d, riBits_h, sizeof(Byte)*N_ri, hipMemcpyHostToDevice, stream_default);
	stopTimer("hipMalloc & hipMemcpy for inputBits & RI_Bits Time= %.6f ms\n", elapsed);

	//Create Plans
	startTimer();
	hipfftHandle plan_transform_precoder;
	int n[1] = { N_sc_rb*M_pusch_rb };
	hipfftPlanMany(&plan_transform_precoder, 1, n, NULL, 1, n[0], NULL, 1, N_sc_rb*M_pusch_rb, HIPFFT_C2C, ((N_bits + N_ri) / Qm)/n[0]);

	hipfftHandle plan_sc_fdma;
	n[0] = { FFT_size };
	hipfftPlanMany(&plan_sc_fdma, 1, n, NULL, 1, FFT_size, NULL, 1, FFT_size, HIPFFT_C2C, N_symbs_per_subframe);
	stopTimer("Create Plans Time= %.6f ms\n", elapsed);

	//Device data allocation
	startTimer();

	//timer_test << <1, 1 >> > ();

	int data_vec_len = Qm*N_l;
	// (UNUSED) int ri_vec_len = Qm*N_l;
	int N_data_bits = N_bits / data_vec_len;
	int N_ri_bits = N_ri / data_vec_len;
	int H_prime = N_data_bits;
	// (UNUSED) int H_vec_len = data_vec_len;
	int H_prime_total = H_prime + N_ri_bits;

	int R_mux = (H_prime_total*Qm*N_l) / N_pusch_symbs;
	int R_prime_mux = R_mux / (Qm*N_l);

	Byte *y_idx_d, *y_mat_d, *interleaved_d;
	hipMalloc((void **)&y_idx_d, sizeof(Byte)*(N_pusch_symbs * R_prime_mux));
	hipMalloc((void **)&y_mat_d, sizeof(Byte)*(N_pusch_symbs*R_mux));
	hipMalloc((void **)&interleaved_d, sizeof(Byte)*(N_pusch_symbs*R_mux));

	Byte *scrambledbits_d = 0;
	hipMalloc((void **)&scrambledbits_d, sizeof(Byte)*N_bits);

	Byte *bits_each_Qm_d;
	//float* symbols_R_d = 0, *symbols_I_d = 0;
	hipMalloc((void **)&bits_each_Qm_d, sizeof(Byte)*(N_bits / Qm));
	//hipMalloc((void **)&symbols_R_d, sizeof(float)*(N_bits / Qm));
	//hipMalloc((void **)&symbols_I_d, sizeof(float)*(N_bits / Qm));

	hipfftComplex *precoded_symbols_d = 0, *cuComplex_symbols_d = 0;
	hipMalloc((void **)&cuComplex_symbols_d, sizeof(hipfftComplex)*(N_bits / Qm));
	hipMalloc((void **)&precoded_symbols_d, sizeof(hipfftComplex)*(N_bits / Qm));

	hipfftComplex* x_q_d;
	hipfftComplex* dmrs_1_d = 0, *dmrs_2_d = 0;
	hipMalloc((void **)&dmrs_1_d, sizeof(hipfftComplex)*N_sc_rb*M_pusch_rb);
	hipMalloc((void **)&dmrs_2_d, sizeof(hipfftComplex)*N_sc_rb*M_pusch_rb);
	hipMalloc((void **)&x_q_d, sizeof(hipfftComplex)*prime_nums[M_pusch_rb - 1]);
	
	hipfftComplex *subframe_d = 0;
	hipMalloc((void **)&subframe_d, sizeof(hipfftComplex)*N_symbs_per_subframe*N_sc_rb*M_pusch_rb);
	
	hipfftComplex* ifft_vec_d;
	hipfftComplex *pusch_bb_d = 0;
	hipMalloc((void **)&ifft_vec_d, sizeof(hipfftComplex)*N_symbs_per_subframe*FFT_size);
	hipMalloc((void **)&pusch_bb_d, sizeof(hipfftComplex)*modulated_subframe_length);
	stopTimer("Device data allocation Time= %.6f ms\n", elapsed);

	/*startTimer();
	stopTimer("Overhead of timer= %.6f ms\n", elapsed);*/
	int times = 1;
	startTimer();
	//Generate Pseudo Random Seq.
	Byte *c_h = 0;
	generate_psuedo_random_seq(&c_h, N_bits, n_RNTI, n_s, N_id_cell);

	//Copy (c) to Device
	hipMemcpyAsync(c_d, c_h, sizeof(Byte)*N_bits, hipMemcpyHostToDevice, stream_default);

	//for (int i = 0; i < times; i++)
	//{
		//Interleaver
		interleaver(inputBits_d, riBits_d, &interleaved_d, N_bits, N_ri, Qm, N_l, y_idx_d, y_mat_d);

		//Generate DMRS
		generate_dmrs_pusch(0, N_id_cell, 0, 0, 0, 0, 0, "fixed", M_pusch_rb, 0, &dmrs_1_d, &dmrs_2_d, x_q_d, stream_dmrs);

		//Scrambler
		scrambler(interleaved_d, &scrambledbits_d, c_d, N_bits + N_ri);

		//Mapper
		mapper(scrambledbits_d, N_bits + N_ri, Qm, M_pusch_rb, cuComplex_symbols_d, bits_each_Qm_d); // Mohammed

		//Transform Precoder
		transform_precoder(&precoded_symbols_d, plan_transform_precoder, cuComplex_symbols_d);
		//Multiplexing the DMRS with the Data
		compose_subframe(precoded_symbols_d, dmrs_1_d, dmrs_2_d, M_pusch_rb, &subframe_d);

		// Generate SC-FDMA signal
		sc_fdma_modulator(subframe_d, M_pusch_rb, &pusch_bb_d, plan_sc_fdma, ifft_vec_d);

		//timer_test << <1, 1 >> > ();
		//startTimer();
		hipfftComplex *pusch_bb_h = (hipfftComplex *)malloc(sizeof(hipfftComplex)*(30720));
		hipMemcpyAsync(pusch_bb_h, pusch_bb_d, sizeof(hipfftComplex)*(30720), hipMemcpyDeviceToHost, stream_default);
	//}
	stopTimer("Processing Time= %.6f ms\n", elapsed/ times);

	//To compare with MATLAB results
	//Run the file (output.m)
	//int NNN = modulated_subframe_length;
	//FILE *results;
	//if ((results = freopen("output.m", "w+", stdout)) == NULL) {
	//	printf("Cannot open file.\n");
	//	exit(1);
	//}

	//printf("clear; clc;\nsymbols_real = [ ");
	//for (int i = 0; i < NNN; i++)
	//{
	//	printf("%10f", pusch_bb_h[i].x);
	//	if (i != (NNN -1))
	//		printf(",");
	//}

	//printf(" ];\nsymbols_imag = [ ");

	//for (int i = 0; i < NNN; i++)
	//{
	//	printf("%10f", pusch_bb_h[i].y);
	//	if (i != (NNN -1))
	//		printf(",");
	//}

	//printf(" ];\n");
	//printf("symbols_CUDA = symbols_real + 1i * symbols_imag;\n");

	////Matlab code
	//printf("matlab_test");

	//fclose(results);

	//if ((results = freopen("matlab_test.m", "w+", stdout)) == NULL) {
	//	printf("Cannot open file.\n");
	//	exit(1);
	//}

	//printf("N_bits = %d; \n", N_bits);
	//if(Qm == 6)
	//	printf("mod_type = %s; \n", "'64qam'");
	//else if (Qm == 4)
	//	printf("mod_type = %s; \n", "'16qam'");
	//else if (Qm == 2)
	//	printf("mod_type = %s; \n", "'qpsk'");
	//else if (Qm == 1)
	//	printf("mod_type = %s; \n", "'bpsk'");
	//
	//printf("N_sc_rb   = 12;      %% number of subcarriers in each resource block\n");
	//printf("M_pusch_rb = %d;      %% number of resource blocks assigned to the UE\n", M_pusch_rb);
	//printf("M_pusch_sc = M_pusch_rb*N_sc_rb;  %% total number of subcarriers\n\n");
	//printf("N_l = %d; \nQ_m = %d; \ndata_bits = (fread(fopen('%s')) - '0').';\nri_bits = (fread(fopen('%s'))-'0').'; \n", N_l, Qm, argv[1], argv[2]);
	//printf("interleaved_bits = channel_interleaver(data_bits, ri_bits, [], Q_m, N_l); \nc_init = 10 * 2 ^ 14 + floor(0 / 2) * 2 ^ 9 + 2; \nc = generate_psuedo_random_seq(c_init, N_bits); \nb_scrampled = scrambler(interleaved_bits, c); \nmapped = mapper(b_scrampled, mod_type); \nprecoded_data = transform_precoder(mapped, M_pusch_rb); \n\ndmrs = generate_dmrs_pusch(0, 2, 0, 0, 0, 0, 0, 'fixed', M_pusch_rb, 0);\ndmrs_1 = dmrs(1:M_pusch_sc);\ndmrs_2 = dmrs(M_pusch_sc+1:2*M_pusch_sc);\nsubframe_1 = compose_subframe(precoded_data, dmrs_1, dmrs_2, M_pusch_rb);\nsymbols_MATLAB = sc_fdma_modulator(subframe_1, M_pusch_rb);\n\nsum((abs(symbols_MATLAB) - abs(symbols_CUDA)))");

	//fclose(results);

	// Free allocated memory
	// free device arrays
	hipFree(inputBits_d);
	hipFree(riBits_d);
	hipFree(c_d);
	hipFree(y_idx_d);
	hipFree(y_mat_d);
	hipFree(interleaved_d);
	hipFree(scrambledbits_d);
	hipFree(bits_each_Qm_d);
	hipFree(cuComplex_symbols_d);
	hipFree(precoded_symbols_d);
	hipFree(dmrs_1_d);
	hipFree(dmrs_2_d);
	hipFree(x_q_d);
	hipFree(subframe_d);
	hipFree(ifft_vec_d);
	hipFree(pusch_bb_d);

	// free host arrays 
	//free(inputBits_h);

}