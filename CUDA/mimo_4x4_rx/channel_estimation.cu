#include "hip/hip_runtime.h"
//  Function:    channe_estimation
//	Description : Generates channel estimation
//	Inputs : symb_0 - received DMRS number 1
//			 symb_1 - received DMRS number 2
//			 dmrs_0 - generated DMRS number 1
//			 dmrs_1 - generated DMRS number 2
//			 M_pusch_sc - number of subcarriers allocated to ue
//	Outputs : channel - channel estimation matrix to be used for equalization
//			  Noise_power : N0
//	edit : 9 / 3 / 2017
//	By : Ahmed Moustafa

#include "channel_estimation.cuh"



__global__ void divide_pilots(hipfftComplex* symb_0_d, hipfftComplex* symb_1_d, hipfftComplex* dmrs_0_d, hipfftComplex* dmrs_1_d, hipfftComplex* filtered_pilots1_d, hipfftComplex* filtered_pilots2_d, const int M_pusch_sc)
{
 	int x_idx = blockIdx.x;
	int y_idx = threadIdx.y + blockIdx.y * blockDim.y ;

	if (y_idx >= M_pusch_sc)
		return;

	// divide the received pilots over the generated reference to get value of the channel 
	if (x_idx == 0)
	{
		filtered_pilots1_d[y_idx] = hipCdivf(symb_0_d[y_idx], dmrs_0_d[y_idx]);
	}
	else
	{
		filtered_pilots2_d[y_idx] = hipCdivf(symb_1_d[y_idx], dmrs_1_d[y_idx]);
	}


	// sync the threads to go  the time average 
	__syncthreads();
	
	if (x_idx != 0)
		return;


	//time average 
	filtered_pilots1_d[y_idx] = (filtered_pilots1_d[y_idx] + filtered_pilots2_d[y_idx] ) / 2;

	// sync the threads for freq average
	__syncthreads();


	// freq avergae with moving average filter of window size 19
	if (y_idx > 8 && y_idx < (M_pusch_sc - 9) )

		filtered_pilots2_d[y_idx] = (filtered_pilots1_d[y_idx - 9] + filtered_pilots1_d[y_idx - 8] + filtered_pilots1_d[y_idx - 7] + filtered_pilots1_d[y_idx - 6] + filtered_pilots1_d[y_idx - 5] + filtered_pilots1_d[y_idx - 4] + filtered_pilots1_d[y_idx - 3] + filtered_pilots1_d[y_idx - 2] + filtered_pilots1_d[y_idx - 1] + filtered_pilots1_d[y_idx] + filtered_pilots1_d[y_idx + 1] + filtered_pilots1_d[y_idx + 2] + filtered_pilots1_d[y_idx + 3] + filtered_pilots1_d[y_idx + 4] + filtered_pilots1_d[y_idx + 5] + filtered_pilots1_d[y_idx + 6] + filtered_pilots1_d[y_idx + 7] + filtered_pilots1_d[y_idx + 8] + filtered_pilots1_d[y_idx + 9]) / 19;

	else if (y_idx == 0 || y_idx == (M_pusch_sc - 1))

		filtered_pilots2_d[y_idx] = filtered_pilots1_d[y_idx];

	else if(y_idx == 1 || y_idx == (M_pusch_sc - 2))

		filtered_pilots2_d[y_idx] = (filtered_pilots1_d[y_idx-1] + filtered_pilots1_d[y_idx] + filtered_pilots1_d[y_idx+1]) / 3;

	else if(y_idx == 2 || y_idx == (M_pusch_sc - 3))

		filtered_pilots2_d[y_idx] = (filtered_pilots1_d[y_idx - 2] + filtered_pilots1_d[y_idx - 1] + filtered_pilots1_d[y_idx] + filtered_pilots1_d[y_idx + 1] + filtered_pilots1_d[y_idx + 2]) / 5;

	else if(y_idx == 3 || y_idx == (M_pusch_sc - 4))

		filtered_pilots2_d[y_idx] = (filtered_pilots1_d[y_idx - 3] + filtered_pilots1_d[y_idx - 2] + filtered_pilots1_d[y_idx - 1] + filtered_pilots1_d[y_idx] + filtered_pilots1_d[y_idx + 1] + filtered_pilots1_d[y_idx + 2] + filtered_pilots1_d[y_idx + 3]) / 7;

	else if(y_idx == 4 || y_idx == (M_pusch_sc - 5))

		filtered_pilots2_d[y_idx] = (filtered_pilots1_d[y_idx - 4] + filtered_pilots1_d[y_idx - 3] + filtered_pilots1_d[y_idx - 2] + filtered_pilots1_d[y_idx - 1] + filtered_pilots1_d[y_idx] + filtered_pilots1_d[y_idx + 1] + filtered_pilots1_d[y_idx + 2] + filtered_pilots1_d[y_idx + 3] + filtered_pilots1_d[y_idx + 4] ) / 9;

	else if(y_idx == 5 || y_idx == (M_pusch_sc - 6))

		filtered_pilots2_d[y_idx] = (filtered_pilots1_d[y_idx - 5] + filtered_pilots1_d[y_idx - 4] + filtered_pilots1_d[y_idx - 3] + filtered_pilots1_d[y_idx - 2] + filtered_pilots1_d[y_idx - 1] + filtered_pilots1_d[y_idx] + filtered_pilots1_d[y_idx + 1] + filtered_pilots1_d[y_idx + 2] + filtered_pilots1_d[y_idx + 3] + filtered_pilots1_d[y_idx + 4] + filtered_pilots1_d[y_idx + 5]) / 11;

	else if(y_idx == 6 || y_idx == (M_pusch_sc - 7))

		filtered_pilots2_d[y_idx] = (filtered_pilots1_d[y_idx - 6] + filtered_pilots1_d[y_idx - 5] + filtered_pilots1_d[y_idx - 4] + filtered_pilots1_d[y_idx - 3] + filtered_pilots1_d[y_idx - 2] + filtered_pilots1_d[y_idx - 1] + filtered_pilots1_d[y_idx] + filtered_pilots1_d[y_idx + 1] + filtered_pilots1_d[y_idx + 2] + filtered_pilots1_d[y_idx + 3] + filtered_pilots1_d[y_idx + 4] + filtered_pilots1_d[y_idx + 5] + filtered_pilots1_d[y_idx + 6]) / 13;

	else if(y_idx == 7 || y_idx == (M_pusch_sc - 8))

		filtered_pilots2_d[y_idx] = (filtered_pilots1_d[y_idx - 7] + filtered_pilots1_d[y_idx - 6] + filtered_pilots1_d[y_idx - 5] + filtered_pilots1_d[y_idx - 4] + filtered_pilots1_d[y_idx - 3] + filtered_pilots1_d[y_idx - 2] + filtered_pilots1_d[y_idx - 1] + filtered_pilots1_d[y_idx] + filtered_pilots1_d[y_idx + 1] + filtered_pilots1_d[y_idx + 2] + filtered_pilots1_d[y_idx + 3] + filtered_pilots1_d[y_idx + 4] + filtered_pilots1_d[y_idx + 5] + filtered_pilots1_d[y_idx + 6] + filtered_pilots1_d[y_idx + 7]) / 15;

	else if(y_idx == 8 || y_idx == (M_pusch_sc - 9))

		filtered_pilots2_d[y_idx] = (filtered_pilots1_d[y_idx - 8] + filtered_pilots1_d[y_idx - 7] + filtered_pilots1_d[y_idx - 6] + filtered_pilots1_d[y_idx - 5] + filtered_pilots1_d[y_idx - 4] + filtered_pilots1_d[y_idx - 3] + filtered_pilots1_d[y_idx - 2] + filtered_pilots1_d[y_idx - 1] + filtered_pilots1_d[y_idx] + filtered_pilots1_d[y_idx + 1] + filtered_pilots1_d[y_idx + 2] + filtered_pilots1_d[y_idx + 3] + filtered_pilots1_d[y_idx + 4] + filtered_pilots1_d[y_idx + 5] + filtered_pilots1_d[y_idx + 6] + filtered_pilots1_d[y_idx + 7] + filtered_pilots1_d[y_idx + 8]) / 17;


	//for mimo

	//// sync the threads for noise estimation
	//__syncthreads();


	////noise estimate
	//filtered_pilots1_d[y_idx] = filtered_pilots1_d[y_idx] - filtered_pilots2_d[y_idx];



}



//void channe_estimation(hipfftComplex* symb_0_h, hipfftComplex* symb_1_h, hipfftComplex* dmrs_0_h, hipfftComplex* dmrs_1_h, const int M_pusch_sc, hipfftComplex** channel_h)
void channe_estimation(hipfftComplex* symb_0_d, hipfftComplex* symb_1_d, hipfftComplex* dmrs_0_d, hipfftComplex* dmrs_1_d, const int M_pusch_sc, hipfftComplex** channel_d)
{
	hipfftComplex* filtered_pilots1_d, *filtered_pilots2_d, *filtered_pilots2_h;
	
	hipMalloc((void **)&filtered_pilots1_d, sizeof(hipfftComplex)*M_pusch_sc);
	hipMalloc((void **)&filtered_pilots2_d, sizeof(hipfftComplex)*M_pusch_sc);
	
	filtered_pilots2_h = (hipfftComplex*) malloc(sizeof(hipfftComplex)*M_pusch_sc);

	//Calling the kernel(s)

	//---------------------------------------step 1 : Determine channel estimates
	int thread_y, grid_y;
	if (M_pusch_sc > 1024)
	{
		thread_y = M_pusch_sc/2;
		grid_y = 2;
	}
	else
	{
		thread_y = M_pusch_sc;
		grid_y = 1;
	}
	dim3 blockDim(1, thread_y);
	dim3 gridDim(2, grid_y);
	
	//startTimer();
	divide_pilots << < gridDim, blockDim >> > (symb_0_d, symb_1_d, dmrs_0_d, dmrs_1_d, filtered_pilots1_d, filtered_pilots2_d, M_pusch_sc);
	//stopTimer("divide_pilots= %.6f ms\n", elapsed);
	
	*channel_d = filtered_pilots2_d;
	//hipMemcpy(filtered_pilots2_h, filtered_pilots2_d, sizeof(hipfftComplex)*M_pusch_sc, hipMemcpyDeviceToHost);
	//FILE *results1;
	//if ((results1 = freopen("channel.m", "w+", stdout)) == NULL) {
	//	printf("Cannot open file.\n");
	//	//exit(1);
	//}

	////input subframe
	//printf("clc;\nx_inside = [ ");
	//for (int i = 0; i < M_pusch_sc; i++)
	//{
	//	printf("%f ", filtered_pilots2_h[i].x);
	//}
	//printf(" ]; ");
	//printf("\n");
	//printf("y_inside = [ ");
	//for (int i = 0; i < M_pusch_sc; i++)
	//{
	//	printf("%f ", filtered_pilots2_h[i].y);
	//}
	//printf(" ];\n ");
	//printf("channel_cuda_inside = x_inside + 1i * y_inside;\nsum(round(channel_cuda_inside,6) - round(channel_cuda,6))");
	//fclose(results1);

	// Cleanup
	/*hipFree(symb_0_d);
	hipFree(symb_1_d);
	hipFree(dmrs_0_d);
	hipFree(dmrs_1_d);
	*///hipFree(filtered_pilots1_d);
	//hipFree(filtered_pilots2_d);
	//hipFree(channel_d);

	//Destroy timers
	//destroyTimers();

}
