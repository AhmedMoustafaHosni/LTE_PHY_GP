/*
% Function:		decompose_subframe
% Description:	compose the subframe by multiplexing the dmrs signal and data
% Inputs:		*subframe_h			the subframe with data of all ofdm symbols
%				M_pusch_rb			number of resource blocks assigned to the ue
% Outputs:		*complex_data_h:	complex data to be sent in subframe
%				*dmrs_1_h:			demodulation reference signal number 1
%				*dmrs_2_h:			demodulation reference signal number 2
By: Ahmad Nour & Mohammed Mostafa
*/

#include "decompose_subframe.cuh"


int main(int argc, char **argv) {

	const int M_pusch_rb = 100;
	const int M_pusch_sc = N_sc_rb * M_pusch_rb;

	//input
	hipfftComplex* subframe_h = (hipfftComplex *)malloc(sizeof(hipfftComplex) * N_symbs_per_subframe * M_pusch_sc);


	for (int i = 0; i < N_symbs_per_subframe * M_pusch_sc; i++)
	{
		subframe_h[i].x = rand() / (float)RAND_MAX * 10;
		subframe_h[i].y = rand() / (float)RAND_MAX * 10;
	}


	//For output
	hipfftComplex *complex_data_h;
	hipfftComplex *dmrs_1_h;
	hipfftComplex *dmrs_2_h;

	//Call the Transform Precoder Function
	decompose_subframe(subframe_h, M_pusch_rb, &complex_data_h, &dmrs_1_h, &dmrs_2_h);

	//Print results
	for (int i = 0; i < 12*M_pusch_sc; i++)
	{
		printf("idx = %d \t %f \t %f \n", i + 1, complex_data_h[i].x, complex_data_h[i].y);
	}

	//input file
	FILE *results;
	if ((results = freopen("decompose_subframe_input.m", "w+", stdout)) == NULL) {
		printf("Cannot open file.\n");
		exit(1);
	}

	printf("clear; clc;\nsymbols_real = [ ");
	for (int i = 0; i < (N_symbs_per_subframe*M_pusch_sc); i++)
	{
		printf("%10f", subframe_h[i].x);
		if (i != ((N_symbs_per_subframe*M_pusch_sc) - 1))
			printf(",");
	}

	printf(" ];\nsymbols_imag = [ ");

	for (int i = 0; i < (N_symbs_per_subframe*M_pusch_sc); i++)
	{
		printf("%10f", subframe_h[i].y);
		if (i != ((N_symbs_per_subframe*M_pusch_sc) - 1))
			printf(",");
	}

	printf(" ];\n");
	printf("subframe_CUDA = symbols_real + 1i * symbols_imag;\n");
	fclose(results);

	//output file
	FILE *results1;
	if ((results1 = freopen("decompose_subframe_Results_.m", "w+", stdout)) == NULL) {
		printf("Cannot open file.\n");
		exit(1);
	}

	//complex_data
	printf("clear; clc;\ncomplex_data_real = [ ");
	for (int i = 0; i < (12 * M_pusch_sc); i++)
	{
		printf("%10f", complex_data_h[i].x);
		if (i != ((12 * M_pusch_sc) - 1))
			printf(",");
	}

	printf(" ];\ncomplex_data_imag = [ ");

	for (int i = 0; i < (12 * M_pusch_sc); i++)
	{
		printf("%10f", complex_data_h[i].y);
		if (i != ((12 * M_pusch_sc) - 1))
			printf(",");
	}

	printf(" ];\n");
	printf("complex_data_CUDA = complex_data_real + 1i * complex_data_imag;\n");

	//dmrs_1
	printf("dmrs1_real = [ ");
	for (int i = 0; i < (M_pusch_sc); i++)
	{
		printf("%10f", dmrs_1_h[i].x);
		if (i != ((M_pusch_sc)-1))
			printf(",");
	}

	printf(" ];\ndmrs1_imag = [ ");

	for (int i = 0; i < (M_pusch_sc); i++)
	{
		printf("%10f", dmrs_1_h[i].y);
		if (i != ((M_pusch_sc)-1))
			printf(",");
	}

	printf(" ];\n");
	printf("dmrs1_CUDA = dmrs1_real + 1i * dmrs1_imag;\n");

	//dmrs2
	printf("dmrs2_real = [ ");
	for (int i = 0; i < (M_pusch_sc); i++)
	{
		printf("%10f", dmrs_2_h[i].x);
		if (i != ((M_pusch_sc)-1))
			printf(",");
	}

	printf(" ];\ndmrs2_imag = [ ");

	for (int i = 0; i < (M_pusch_sc); i++)
	{
		printf("%10f", dmrs_2_h[i].y);
		if (i != ((M_pusch_sc)-1))
			printf(",");
	}

	printf(" ];\n");
	printf("dmrs2_CUDA = dmrs2_real + 1i * dmrs2_imag;\n");

	//close input file
	fclose(results1);

	return 0;

}