#include "hip/hip_runtime.h"
/*
% Function:		compose_subframe
% Description:	compose the subframe by multiplexing the dmrs signal and data
% Inputs:		*complex_data_h:	complex data to be sent in subframe
%				*dmrs_1_h:			demodulation reference signal number 1
%				*dmrs_2_h:			demodulation reference signal number 2
%				M_pusch_rb			number of resource blocks assigned to the ue
% Outputs:		*subframe_h			the subframe with data of all ofdm symbols
By: Ahmad Nour & Mohammed Mostafa
*/

/*
coeff_multiply kernel just multiples the output symbols by a coeff. The kernel's overhead can be avoided if we
merged it with the mapper kernel
*/

#include "compose_subframe.cuh"

__global__ void compose_subframe(hipfftComplex* complex_data_d, hipfftComplex* dmrs_1_d, hipfftComplex* dmrs_2_d, int M_pusch_sc, hipfftComplex* subframe_d) {

	int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int y_idx = blockIdx.y;

	if (y_idx == 3)
		subframe_d[y_idx*M_pusch_sc + x_idx] = dmrs_1_d[x_idx];
	else if (y_idx == 10)
		subframe_d[y_idx*M_pusch_sc + x_idx] = dmrs_2_d[x_idx];
	else if (y_idx > 10)
		subframe_d[y_idx*M_pusch_sc + x_idx] = complex_data_d[(y_idx - 2)*M_pusch_sc + x_idx];
	else if (y_idx > 3)
		subframe_d[y_idx*M_pusch_sc + x_idx] = complex_data_d[(y_idx - 1)*M_pusch_sc + x_idx];
	else
		subframe_d[y_idx*M_pusch_sc + x_idx] = complex_data_d[y_idx*M_pusch_sc + x_idx];

}



void compose_subframe(hipfftComplex* complex_data_d, hipfftComplex* dmrs_1_d, hipfftComplex* dmrs_2_d, const int M_pusch_rb, hipfftComplex** subframe_d)
{
	int M_pusch_sc = N_sc_rb * M_pusch_rb;

	//compose subframe
	dim3 grid(2, N_symbs_per_subframe, 1);
	dim3 block(M_pusch_sc / 2, 1, 1);
	compose_subframe << < grid, block >> >(complex_data_d, dmrs_1_d, dmrs_2_d, M_pusch_sc, *subframe_d);

}