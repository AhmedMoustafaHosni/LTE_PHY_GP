#include "hip/hip_runtime.h"
/*

Function:		 channel_equalization_zf
Description : equalise the channel effecct on the received signal


Inputs :	  modulated_subframe - received subframe without demodulation signal in one vector
			  channel - estimated channel


Outputs :    equalised symbols

// By : Ahmed Moustafa

*/

#include "channel_equalization_zf.cuh"

__global__ void channel_equalization_zf(hipfftComplex* subframe_d, hipfftComplex*  channel_d, hipfftComplex*  equalized_subframe_d, const int M_pusch_sc)
{
	int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int y_idx = blockIdx.y;

	equalized_subframe_d[y_idx*M_pusch_sc + x_idx] = hipCdivf(subframe_d[y_idx*M_pusch_sc + x_idx], channel_d[x_idx]);
}



void channel_equalization_zf(hipfftComplex* subframe_h, const int M_pusch_sc, hipfftComplex* channel_h, hipfftComplex** equalized_subframe_h)
{
	//For timing purpose
	float elapsed = 0;				//For time calc.
	hipEvent_t start, stop;

	//Device data
	hipfftComplex * subframe_d, *channel_d, *equalized_subframe_d;


	//Host data allocation
	*equalized_subframe_h = (hipfftComplex *)malloc(sizeof(hipfftComplex)*(M_pusch_sc*N_data_symbs_per_subframe));


	//Device data allocation
	startTimer();
	hipMalloc((void **)&subframe_d, sizeof(hipfftComplex)*M_pusch_sc*N_data_symbs_per_subframe);
	hipMalloc((void **)&channel_d, sizeof(hipfftComplex)*M_pusch_sc);
	hipMalloc((void **)&equalized_subframe_d, sizeof(hipfftComplex)*(M_pusch_sc*N_data_symbs_per_subframe));
	stopTimer("hipMalloc Time= %.6f ms\n", elapsed);

	//Copying data to device
	startTimer();
	hipMemcpy(subframe_d, subframe_h, sizeof(hipfftComplex)*M_pusch_sc*N_data_symbs_per_subframe, hipMemcpyHostToDevice);
	hipMemcpy(channel_d, channel_h, sizeof(hipfftComplex)*M_pusch_sc, hipMemcpyHostToDevice);
	stopTimer("hipMemcpy Host->Device Time= %.6f ms\n", elapsed);


	//Calling the kernel(s)

	
	dim3 grid(2, N_data_symbs_per_subframe, 1);
	dim3 block(M_pusch_sc / 2, 1, 1);
	startTimer();
	channel_equalization_zf << < grid, block >> > (subframe_d, channel_d, equalized_subframe_d, M_pusch_sc);
	stopTimer("channel equalization= %.6f ms\n", elapsed);



	hipDeviceSynchronize();


	//Retrieve data from device
	startTimer();
	hipMemcpy(*equalized_subframe_h, equalized_subframe_d, sizeof(hipfftComplex)*M_pusch_sc*N_data_symbs_per_subframe, hipMemcpyDeviceToHost);
	stopTimer("hipMemcpy Device->Host Time= %.6f ms\n", elapsed);


	// Cleanup
	hipFree(subframe_d);
	hipFree(channel_d);
	hipFree(equalized_subframe_d);


	//Destroy timers
	destroyTimers();

}
