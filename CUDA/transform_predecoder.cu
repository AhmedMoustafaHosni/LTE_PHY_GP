#include "hip/hip_runtime.h"
/*
% Function:		transform_predecoder
% Description:	perform transform predecoding on complex data after sc-fdma demdulation
% Inputs:		*symbols_h:			complex data output from sc-fdma demdulator
%				M_pusch_rb			numer of resource blocks assigned to ue
% Outputs:		*prdecoded_data	transform predecodded data
By: Ahmad Nour & Mohammed Mostafa
*/

/*
coeff_multiply kernel just multiples the output symbols by a coeff. The kernel's overhead can be avoided if we
merged it with the mapper kernel
*/

#include "transform_predecoder.cuh"

__global__ void coeff_multiply(hipfftComplex* symbols_d, double coeff, int numThreads) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//Not to run more threads than available data
	if (idx >= numThreads)
		return;

	symbols_d[idx].x *= coeff;
	symbols_d[idx].y *= coeff;
}

void transform_predecoder(hipfftComplex* symbols_h, const int M_pusch_rb, int signal_size, hipfftComplex** precoded_data_h)
{
	int M_pusch_sc = N_sc_rb * M_pusch_rb;

	//For timing purpose
	float elapsed = 0;				//For time calc.
	hipEvent_t start, stop;

	//Device data
	hipfftComplex* symbols_d;
	hipfftComplex* predecoded_data_d;

	//Host data allocation
	*precoded_data_h = (hipfftComplex *)malloc(sizeof(hipfftComplex)*signal_size);
	
	//Device data allocation
	startTimer();
	hipMalloc((void **)&symbols_d, sizeof(hipfftComplex)*signal_size);
	hipMalloc((void **)&predecoded_data_d, sizeof(hipfftComplex)*signal_size);
	stopTimer("hipMalloc Time= %.6f ms\n", elapsed);

	//Copying data to device
	startTimer();
	hipMemcpy(symbols_d, symbols_h, sizeof(hipfftComplex)*signal_size, hipMemcpyHostToDevice);
	stopTimer("hipMemcpy Host->Device Time= %.6f ms\n", elapsed);

	// CUFFT plan
	int N_SIGS = signal_size / M_pusch_sc;
	int n[1] = { M_pusch_sc };
	hipfftHandle plan;

	hipfftPlanMany(&plan, 1, n, NULL, 1, M_pusch_sc, NULL, 1, M_pusch_sc, HIPFFT_C2C, N_SIGS);
	hipfftExecC2C(plan, symbols_d, predecoded_data_d, HIPFFT_BACKWARD);

	//Calc. number of needed threads for calling kernel(s)
	int numThreads = signal_size;
	int blockDim = (numThreads < 1024) ? numThreads : 1024;	//block size in threads (max 1024 thread)
	int gridDim = numThreads / (blockDim)+(numThreads % blockDim == 0 ? 0 : 1); //grid size in bloack (min 1)

	//Coeff. Multiplication
	coeff_multiply << <gridDim, blockDim >> > (predecoded_data_d, rsqrtf(M_pusch_sc), numThreads);

	//Retrieve data from device
	startTimer();
	hipMemcpy(*precoded_data_h, predecoded_data_d, sizeof(hipfftComplex)*signal_size, hipMemcpyDeviceToHost);
	stopTimer("hipMemcpy Device->Host Time= %.6f ms\n", elapsed);

	// Cleanup
	hipFree(symbols_d);
	hipFree(predecoded_data_d);

	//Destroy timers
	destroyTimers();
}
