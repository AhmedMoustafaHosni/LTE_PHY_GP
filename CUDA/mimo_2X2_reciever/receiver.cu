#include "hip/hip_runtime.h"
/*
% Function:    receiver
By: Mohammed Osama & Khaled Ahmed 
*/

#include "sc_fdma_demodulator.cuh"
#include "generate_dmrs_pusch.cuh"
#include "generate_ul_rs.cuh"
#include "generate_psuedo_random_seq.cuh"
#include "transform_predecoder.cuh"
#include "decompose_subframe.cuh"
#include "demapper.cuh"
#include "descrambler.cuh"
#include "deinterleaver.cuh"
#include "channel_estimation.cuh"
#include "channel_equalization_zf.cuh"


int main(int argc, char **argv) {

	//input
	hipfftComplex* subframe_h = (hipfftComplex *)malloc(sizeof(hipfftComplex)*modulated_subframe_length);
	hipfftComplex* subframe_h2 = (hipfftComplex *)malloc(sizeof(hipfftComplex)*modulated_subframe_length);

	for (int i = 0; i < modulated_subframe_length; i++)
	{
		subframe_h[i].x = rand() / (float)RAND_MAX * 0.5;
		subframe_h[i].y = rand() / (float)RAND_MAX * 0.5;
	}

	for (int i = 0; i < modulated_subframe_length; i++)
	{
		subframe_h2[i].x = rand() / (float)RAND_MAX * 0.5;
		subframe_h2[i].y = rand() / (float)RAND_MAX * 0.5;
	}


	//For timing purpose
	timerInit();
	startTimer();

	const int Qm = 6;				// Modulation Order(2 = QPSK, 4 = 16QAM, 6 = 64QAM)
	const int M_pusch_rb = 100;		//number of resource blocks assigned to the UE
	const int N_l = 2;				// Number of Layers
	const int N_ri = 0;			//length of ri symbols
	const int n_s = 0;				//assume UE send on subframe 0
	const int N_id_cell = 2;		//assume enodeB scheduled cell 2 for the UE
	const int M_pusch_sc = N_sc_rb * M_pusch_rb; //total number of subcarriers
	const int n_RNTI = 10;			//radio network temporary identifier given to the UE by enodeB (assume 10)
	const int N_bits = Qm * 12 * M_pusch_sc;   //Qm * 12 * M_pusch_sc = 2*12*1200

	hipfftComplex* subframe_d;
	hipfftComplex* subframe_d2;
	hipMalloc((void **)&subframe_d, sizeof(hipfftComplex)*modulated_subframe_length);
	hipMalloc((void **)&subframe_d2, sizeof(hipfftComplex)*modulated_subframe_length);
	hipMemcpy(subframe_d, subframe_h, sizeof(hipfftComplex)*modulated_subframe_length, hipMemcpyHostToDevice);
	hipMemcpy(subframe_d2, subframe_h2, sizeof(hipfftComplex)*modulated_subframe_length, hipMemcpyHostToDevice);

	//Generate Pseudo Random Seq.
	Byte *c_h = 0;
	generate_psuedo_random_seq(&c_h, N_bits, n_RNTI, n_s, N_id_cell);
	//Copy (c) to Device
	Byte* c_d = 0;
	hipMalloc((void **)&c_d, sizeof(Byte)*Qm * 12 * M_pusch_sc);
	hipMemcpyAsync(c_d, c_h, sizeof(Byte)*N_bits, hipMemcpyHostToDevice);
	stopTimer("Time of copying of data to device= %.6f ms\n", elapsed);


	startTimer();
	//Device data allocation
	hipfftComplex* fft_vec_d;
	hipfftComplex* fft_vec_d2;
	hipfftComplex* demod_subframe_d;
	hipfftComplex* demod_subframe_d2;
	hipfftComplex* demod_subframe_h = (hipfftComplex*)malloc(sizeof(hipfftComplex)*N_symbs_per_subframe*M_pusch_sc);
	hipfftComplex* demod_subframe_h2 = (hipfftComplex*)malloc(sizeof(hipfftComplex)*N_symbs_per_subframe*M_pusch_sc);

	hipMalloc((void **)&fft_vec_d, sizeof(hipfftComplex)*N_symbs_per_subframe*FFT_size);
	hipMalloc((void **)&fft_vec_d2, sizeof(hipfftComplex)*N_symbs_per_subframe*FFT_size);
	hipMalloc((void **)&demod_subframe_d, sizeof(hipfftComplex)*N_symbs_per_subframe*M_pusch_sc);
	hipMalloc((void **)&demod_subframe_d2, sizeof(hipfftComplex)*N_symbs_per_subframe*M_pusch_sc);

	hipfftComplex* x_q_d;
	hipfftComplex* x_q_d2;
	hipfftComplex* dmrs1_generated_d = 0, *dmrs2_generated_d = 0;
	hipfftComplex* dmrs1_generated_d2 = 0, *dmrs2_generated_d2 = 0;
	hipMalloc((void **)&dmrs1_generated_d, sizeof(hipfftComplex)*N_sc_rb*M_pusch_rb);
	hipMalloc((void **)&dmrs2_generated_d, sizeof(hipfftComplex)*N_sc_rb*M_pusch_rb);
	hipMalloc((void **)&dmrs1_generated_d2, sizeof(hipfftComplex)*N_sc_rb*M_pusch_rb);
	hipMalloc((void **)&dmrs2_generated_d2, sizeof(hipfftComplex)*N_sc_rb*M_pusch_rb);

	hipMalloc((void **)&x_q_d, sizeof(hipfftComplex)*prime_nums[M_pusch_rb - 1]);
	hipMalloc((void **)&x_q_d2, sizeof(hipfftComplex)*prime_nums[M_pusch_rb - 1]);

	hipfftComplex* dmrs1_decomposed_d;
	hipfftComplex* dmrs2_decomposed_d;
	hipfftComplex* dmrs1_decomposed_d2;
	hipfftComplex* dmrs2_decomposed_d2;
	hipfftComplex* complex_data_d;
	hipfftComplex* complex_data_d2;
	hipfftComplex* complex_data_h = (hipfftComplex*)malloc(sizeof(hipfftComplex)* 12 * M_pusch_sc);
	hipfftComplex* complex_data_h2 = (hipfftComplex*)malloc(sizeof(hipfftComplex)* 12 * M_pusch_sc);
	hipMalloc((void **)&complex_data_d, sizeof(hipfftComplex)* 12 * M_pusch_sc);
	hipMalloc((void **)&complex_data_d2, sizeof(hipfftComplex)* 12 * M_pusch_sc);
	hipMalloc((void **)&dmrs1_decomposed_d, sizeof(hipfftComplex)*M_pusch_sc);
	hipMalloc((void **)&dmrs1_decomposed_d2, sizeof(hipfftComplex)*M_pusch_sc);
	hipMalloc((void **)&dmrs2_decomposed_d, sizeof(hipfftComplex)*M_pusch_sc);
	hipMalloc((void **)&dmrs2_decomposed_d2, sizeof(hipfftComplex)*M_pusch_sc);


	// Channel estimation and equaliuzation allocation
	/*hipfftComplex* channel, *equalized_subframe_d;
	hipMalloc((void **)&channel, sizeof(hipfftComplex)*M_pusch_sc);
	hipMalloc((void **)&equalized_subframe_d, sizeof(hipfftComplex)* 12 * M_pusch_sc);

	hipfftComplex* equalized_subframe_h = (hipfftComplex*)malloc(sizeof(hipfftComplex)* 12 * M_pusch_sc);

	hipfftComplex* channel_h = (hipfftComplex*)malloc(sizeof(hipfftComplex)*M_pusch_sc);*/


	hipfftComplex* predecoded_data_d;
	hipMalloc((void **)&predecoded_data_d, sizeof(hipfftComplex)* 12 * M_pusch_sc);
	hipfftComplex* predecoded_data_d2;
	hipMalloc((void **)&predecoded_data_d2, sizeof(hipfftComplex)* 12 * M_pusch_sc);

	Byte *bits_d;
	hipMalloc((void **)&bits_d, sizeof(Byte)* Qm * 12 * M_pusch_sc);    //FIX Number_demaped_bits
	Byte *bits_d2;
	hipMalloc((void **)&bits_d2, sizeof(Byte)* Qm * 12 * M_pusch_sc);    //FIX Number_demaped_bits

	Byte *descrambled_bits_d;
	hipMalloc((void **)&descrambled_bits_d, sizeof(Byte)* Qm * 12 * M_pusch_sc);
	Byte *descrambled_bits_d2;
	hipMalloc((void **)&descrambled_bits_d2, sizeof(Byte)* Qm * 12 * M_pusch_sc);
	Byte *descrambled_bits_h = (Byte *)malloc(sizeof(Byte)* Qm * 12 * M_pusch_sc);
	Byte *descrambled_bits_h2 = (Byte *)malloc(sizeof(Byte)* Qm * 12 * M_pusch_sc);


	// Step 1: Define C_mux
	int C_mux = N_pusch_symbs;
	// Step 2: Define R_mux and R_prime_mux
	int H_prime_total = N_bits * N_l / (Qm*N_l);
	int H_prime = H_prime_total - N_ri;
	int R_mux = (H_prime_total*Qm*N_l) / C_mux;
	int R_prime_mux = R_mux / (Qm*N_l);

	Byte *ri_d, *y_idx_d, *y_mat_d;
	Byte *received_bits_d;
	//Byte *received_bits_h = (Byte *)malloc(sizeof(Byte *) * N_bits);
	hipMalloc((void **)&ri_d, sizeof(Byte)*(N_ri * Qm * N_l));
	hipMalloc((void **)&y_idx_d, sizeof(Byte)*(C_mux*R_prime_mux));
	hipMalloc((void **)&y_mat_d, sizeof(Byte)*(C_mux*R_mux));
	hipMalloc((void **)&received_bits_d, sizeof(Byte)* H_prime * Qm * N_l);
	stopTimer("Allocation Time= %.6f ms\n", elapsed);

	startTimer();
	//create plans
	int n[1] = { FFT_size };
	hipfftHandle plan_sc_fdma, plan_sc_fdma2;
	hipfftPlanMany(&plan_sc_fdma, 1, n, NULL, 1, FFT_size, NULL, 1, FFT_size, HIPFFT_C2C, N_symbs_per_subframe);
	hipfftPlanMany(&plan_sc_fdma2, 1, n, NULL, 1, FFT_size, NULL, 1, FFT_size, HIPFFT_C2C, N_symbs_per_subframe);

	int N_SIGS = 12;   //signal_size/M_pusch_sc = 12 * M_pusch_sc / M_pusch_sc = 12
	n[0] = { M_pusch_sc };
	hipfftHandle plan_transform_predecoder;
	hipfftHandle plan_transform_predecoder2;
	hipfftPlanMany(&plan_transform_predecoder, 1, n, NULL, 1, M_pusch_sc, NULL, 1, M_pusch_sc, HIPFFT_C2C, N_SIGS);
	hipfftPlanMany(&plan_transform_predecoder2, 1, n, NULL, 1, M_pusch_sc, NULL, 1, M_pusch_sc, HIPFFT_C2C, N_SIGS);
	stopTimer("Time of plan creation= %.6f ms\n", elapsed);

	startTimer();

	//sc-fdma demodulation
	sc_fdma_demodulator(subframe_d, M_pusch_rb, &demod_subframe_d, plan_sc_fdma, fft_vec_d);
	sc_fdma_demodulator(subframe_d2, M_pusch_rb, &demod_subframe_d2, plan_sc_fdma2, fft_vec_d2);

	//hipMemcpy(demod_subframe_h, demod_subframe_d, sizeof(hipfftComplex)*N_symbs_per_subframe*M_pusch_sc, hipMemcpyDeviceToHost);
	//hipMemcpy(demod_subframe_h2, demod_subframe_d2, sizeof(hipfftComplex)*N_symbs_per_subframe*M_pusch_sc, hipMemcpyDeviceToHost);
	//generate dmrs   
	//generate_dmrs_pusch(n_s, N_id_cell, 0, 0, 0, 0, 0, "fixed", M_pusch_rb, 0, &dmrs1_generated_d, &dmrs2_generated_d, x_q_d);
	//generate_dmrs_pusch(n_s, N_id_cell, 0, 0, 0, 0, 0, "fixed", M_pusch_rb, 1, &dmrs1_generated_d2, &dmrs2_generated_d2, x_q_d2);


	//Decompose subframe
	decompose_subframe(demod_subframe_d, M_pusch_rb, &complex_data_d, &dmrs1_decomposed_d, &dmrs2_decomposed_d);
	decompose_subframe(demod_subframe_d2, M_pusch_rb, &complex_data_d2, &dmrs1_decomposed_d2, &dmrs2_decomposed_d2);

	//Channel estimation
	//channe_estimation(dmrs1_decomposed_d, dmrs2_decomposed_d, dmrs1_generated_d, dmrs2_generated_d, M_pusch_sc, &channel);

	//hipMemcpy(channel_h, channel, sizeof(hipfftComplex)* M_pusch_sc, hipMemcpyDeviceToHost);

	//Equalization ZF
	//channel_equalization_zf(demod_subframe_d, M_pusch_sc , channel, &equalized_subframe_d);

	//hipMemcpy(complex_data_h, complex_data_d, sizeof(hipfftComplex)* 12 * M_pusch_sc, hipMemcpyDeviceToHost);
	//hipMemcpy(complex_data_h2, complex_data_d2, sizeof(hipfftComplex)* 12 * M_pusch_sc, hipMemcpyDeviceToHost);

	//predecoding   
	transform_predecoder(complex_data_d, &predecoded_data_d, plan_transform_predecoder); //signal_size = 12 * M_pusch_sc
	transform_predecoder(complex_data_d2, &predecoded_data_d2, plan_transform_predecoder2); //signal_size = 12 * M_pusch_sc

	//demapping
	demapper(predecoded_data_d, M_pusch_rb, &bits_d, Qm * 12 * M_pusch_sc, Qm);  //Number_demaped_bits = Qm * 12 * M_pusch_sc
	demapper(predecoded_data_d2, M_pusch_rb, &bits_d2, Qm * 12 * M_pusch_sc, Qm);  //Number_demaped_bits = Qm * 12 * M_pusch_sc


	//Descrammpling
	descrambler(bits_d, &descrambled_bits_d, c_d, N_bits);
	descrambler(bits_d2, &descrambled_bits_d2, c_d, N_bits);
	
	//hipMemcpy(descrambled_bits_h, descrambled_bits_d, sizeof(Byte) * N_bits, hipMemcpyDeviceToHost);
	//hipMemcpy(descrambled_bits_h2, descrambled_bits_d2, sizeof(Byte) * N_bits, hipMemcpyDeviceToHost);
	
	//deinterleaver
	deinterleaver(descrambled_bits_d, descrambled_bits_d2, &ri_d, &received_bits_d, N_bits*N_l, N_ri, Qm, N_l, y_idx_d, y_mat_d);

	//hipMemcpy(received_bits_h, received_bits_d, sizeof(Byte *) * N_bits * 2, hipMemcpyDeviceToHost);

	//Retrieve data from device
	Byte* received_bits_h = (Byte*)malloc(sizeof(Byte)*N_bits * N_l);
	hipMemcpy(received_bits_h, received_bits_d, sizeof(Byte)*N_bits* N_l, hipMemcpyDeviceToHost);

	Byte* ri_h = (Byte*)malloc(sizeof(Byte)*N_ri * Qm * N_l);
	hipMemcpy(ri_h, ri_d, sizeof(Byte)*N_ri * Qm * N_l, hipMemcpyDeviceToHost);
	stopTimer("Time of processing= %.6f ms\n", elapsed);

	//Print results
	/*for (int i = 0; i < H_prime * Qm * N_l; i++)
	{
	printf("idx = %d \t %d  \n", i + 1, received_bits_h[i]);
	}
	*/

	//test file
	FILE *results1;
	if ((results1 = freopen("Receiver_test.m", "w+", stdout)) == NULL) {
		printf("Cannot open file.\n");
		exit(1);
	}

	//input subframe
	printf("clear; clc;\nsymbols_in_real = [ ");
	for (int i = 0; i < (modulated_subframe_length); i++)
	{
		printf("%10f", subframe_h[i].x);
		if (i != ((modulated_subframe_length)-1))
			printf(",");
	}
	printf(" ];\nsymbols_in_imag = [ ");
	for (int i = 0; i < (modulated_subframe_length); i++)
	{
		printf("%10f", subframe_h[i].y);
		if (i != ((modulated_subframe_length)-1))
			printf(",");
	}
	printf(" ];\n");
	printf("subframe_CUDA = symbols_in_real + 1i * symbols_in_imag;\n");

	printf("\nsymbols_in_real = [ ");
	for (int i = 0; i < (modulated_subframe_length); i++)
	{
		printf("%10f", subframe_h2[i].x);
		if (i != ((modulated_subframe_length)-1))
			printf(",");
	}
	printf(" ];\nsymbols_in_imag = [ ");
	for (int i = 0; i < (modulated_subframe_length); i++)
	{
		printf("%10f", subframe_h2[i].y);
		if (i != ((modulated_subframe_length)-1))
			printf(",");
	}
	printf(" ];\n");
	printf("subframe_input2_CUDA = symbols_in_real + 1i * symbols_in_imag;\n");
	printf("subframe_input_CUDA = [ subframe_CUDA; subframe_input2_CUDA];\n");

	//printf("subframe_input_CUDA = symbols_in_real + 1i * symbols_in_imag;\n");



	//// Channel estimation 
	//printf("x = [ ");
	//for (int i = 0; i <  M_pusch_sc; i++)
	//{
	//	printf("%f ", channel_h[i].x);
	//}
	//printf(" ]; ");
	//printf("\n");
	//printf("y = [ ");
	//for (int i = 0; i <  M_pusch_sc; i++)
	//{
	//	printf("%f ", channel_h[i].y);
	//}
	//printf(" ];\n ");
	//printf("channel_cuda = x + 1i * y;\n");
	// channel equalization 
	/*printf("x = [ ");
	for (int i = 0; i < (M_pusch_sc*N_data_symbs_per_subframe); i++)
	{
	printf("%f ", equalized_subframe_h[i].x);
	}
	printf(" ]; ");
	printf("\n");
	printf("y = [ ");
	for (int i = 0; i < (M_pusch_sc*N_data_symbs_per_subframe); i++)
	{
	printf("%f ", equalized_subframe_h[i].y);
	}
	printf(" ];\n ");
	printf("equalized_subframe_h = x + 1i * y;\n");*/

	// sc-fdma_demodulation 

	printf("x = [ ");
	for (int i = 0; i < (N_symbs_per_subframe*M_pusch_sc); i++)
	{
		printf("%f ", demod_subframe_h[i].x);
	}
	printf(" ]; ");
	printf("\n");
	printf("y = [ ");
	for (int i = 0; i < (N_symbs_per_subframe*M_pusch_sc); i++)
	{
		printf("%f ", demod_subframe_h[i].y);
	}
	printf(" ];\n ");
	printf("demod_subframe_h = x + 1i * y;\n");

	printf("x = [ ");
	for (int i = 0; i < (N_symbs_per_subframe*M_pusch_sc); i++)
	{
		printf("%f ", demod_subframe_h2[i].x);
	}
	printf(" ]; ");
	printf("\n");
	printf("y = [ ");
	for (int i = 0; i < (N_symbs_per_subframe*M_pusch_sc); i++)
	{
		printf("%f ", demod_subframe_h2[i].y);
	}
	printf(" ];\n ");
	printf("demod_subframe_h2 = x + 1i * y;\n");

	// test decompose subfram 
	printf("x = [ ");
	for (int i = 0; i < (M_pusch_sc*N_data_symbs_per_subframe); i++)
	{
		printf("%f ", complex_data_h[i].x);
	}
	printf(" ]; ");
	printf("\n");
	printf("y = [ ");
	for (int i = 0; i < (M_pusch_sc*N_data_symbs_per_subframe); i++)
	{
		printf("%f ", complex_data_h[i].y);
	}
	printf(" ];\n ");
	printf("complex_data_h = x + 1i * y;\n");

	printf("x = [ ");
	for (int i = 0; i < (M_pusch_sc*N_data_symbs_per_subframe); i++)
	{
		printf("%f ", complex_data_h2[i].x);
	}
	printf(" ]; ");
	printf("\n");
	printf("y = [ ");
	for (int i = 0; i < (M_pusch_sc*N_data_symbs_per_subframe); i++)
	{
		printf("%f ", complex_data_h2[i].y);
	}
	printf(" ];\n ");
	printf("complex_data_h2 = x + 1i * y;\n");

	//Received Bits
	printf("\ndescrambled_bits_cuda = [ ");
	for (int i = 0; i < (N_bits); i++)
	{
		printf("%d", descrambled_bits_h[i]);
		if (i != ((Qm * 12 * M_pusch_sc) - 1))
			printf(",");
	}
	printf(" ];\n");

	printf("\ndescrambled_bits2_cuda = [ ");
	for (int i = 0; i < (N_bits); i++)
	{
		printf("%d", descrambled_bits_h2[i]);
		if (i != ((Qm * 12 * M_pusch_sc) - 1))
			printf(",");
	}
	printf(" ];\n");


	//Received Bits
	printf("\nReceved_bits_cuda = [ ");
	for (int i = 0; i < (N_bits * N_l); i++)
	{
		printf("%d", received_bits_h[i]);
		if (i != ((Qm * 12 * M_pusch_sc * N_l) - 1))
			printf(",");
	}
	printf(" ];\n");



	//RI Bits
	printf("\nRI_bits_cuda = [ ");
	for (int i = 0; i < (N_ri * Qm * N_l); i++)
	{
		printf("%d", ri_h[i]);
		if (i != ((N_ri * Qm * N_l) - 1))
			printf(",");
	}
	printf(" ];\n");

	//printf("N_id_cell = 2;N_sc_rb   = 12;M_pusch_rb = 100;M_pusch_sc = M_pusch_rb*N_sc_rb;Nc = 1600;n_s = 0;n_RNTI = 10;M_bits = 86400;N_l = 2;\nN_ri_bits = 0;N_ack_bits =0;Q_m = 6;\nmodulated_subframe = subframe_input_CUDA;\ndemodulated_subframe = sc_fdma_demodulator(modulated_subframe, M_pusch_rb);\ndemodulated_subframe_vect =[demodulated_subframe(0+1,:), demodulated_subframe(1+1,:), demodulated_subframe(2+1,:), demodulated_subframe(4+1,:), demodulated_subframe(5+1,:), demodulated_subframe(6+1,:), demodulated_subframe(7+1,:), demodulated_subframe(8+1,:), demodulated_subframe(9+1,:), demodulated_subframe(11+1,:), demodulated_subframe(12+1,:), demodulated_subframe(13+1,:)];\ndmrs = generate_dmrs_pusch(n_s, N_id_cell, 0, 0, 0, 0, 0, 'fixed', M_pusch_rb, 0);\ndmrs_1 = dmrs(1:M_pusch_sc);\ndmrs_2 = dmrs(M_pusch_sc+1:2*M_pusch_sc);\ndmrs_1_rx = demodulated_subframe(1+3,:);\ndmrs_2_rx = demodulated_subframe(1+10,:);\npredecoded_data = transform_predecoder(demodulated_subframe_vect, M_pusch_rb);\n demapped_data = demapper_hard(predecoded_data, '64qam');\n c_init = n_RNTI * 2 ^ 14 + floor(n_s / 2) * 2 ^ 9 + N_id_cell;\n c = generate_psuedo_random_seq(c_init, M_bits);\n descrambled_bits = descramble(demapped_data, c);\n [data_bits, ri_bits, ack_bits] = channel_deinterleaver(descrambled_bits, N_ri_bits, N_ack_bits, N_l, Q_m); \nisequal(data_bits, Receved_bits_cuda)\nisequal(ri_bits, RI_bits_cuda)\n");
	printf("N_id_cell = 2;N_sc_rb   = 12;M_pusch_rb = 100;M_pusch_sc = M_pusch_rb*N_sc_rb;Nc = 1600;n_s = 0;n_RNTI = 10;M_bits = 86400;N_l = 2;\nN_ri_bits = 0;N_ack_bits =0;Q_m = 6;\nmodulated_subframe = subframe_input_CUDA;\ndemodulated_subframe = sc_fdma_demodulator_MIMO(modulated_subframe, M_pusch_rb, N_l);\nDecoded_streams = decompose_subframe_mimo(demodulated_subframe, N_l);\ntransform_predecoded_symbols = transform_predecoder_mimo(Decoded_streams.', M_pusch_rb, N_l);\n [layer1, layer2] = layer_demapping(transform_predecoded_symbols, N_l);\ndemapped_bits1 = transpose(demapper_hard(layer1, '64qam'));\ndemapped_bits2 = transpose(demapper_hard(layer2, '64qam'));\n c_init = n_RNTI * 2 ^ 14 + floor(n_s / 2) * 2 ^ 9 + N_id_cell;\n c = generate_psuedo_random_seq(c_init, M_bits);\n descrambled_bits = descrambler_MIMO([demapped_bits1 demapped_bits2].', [c; c], N_l);\n [data_bits, ri_bits, ack_bits] = channel_deinterleaver_MIMO(descrambled_bits.', 0, 0, N_l, Q_m); \nisequal(data_bits, Receved_bits_cuda)\nisequal(ri_bits, RI_bits_cuda)\n");
	//printf("sum(round(complex_data_h,6)-round(Decoded_streams(1,:),6))\n");
	//printf("sum(round(complex_data_h2,6)-round(Decoded_streams(2,:),6))\n");
	/*printf("isequal(descrambled_bits_cuda, descrambled_bits(1,:))\n");
	printf("isequal(descrambled_bits2_cuda, descrambled_bits(2,:))");*/

//	printf("sum(round(Receved_bits_cuda,6)-round(,6))\n");

	fclose(results1);
	return 0;
}