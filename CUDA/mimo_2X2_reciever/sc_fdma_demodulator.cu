#include "hip/hip_runtime.h"
/*
% Function:		sc_fdma_demodulator
% Description:	Generates complex symbols from the sc-fdma symbols
% Inputs		*pusch_bb_h		sc-fdma symbols
M_pusch_rb		numer of resource blocks assigned to ue
% Outputs:		*symbs_h		output symbols
By: Mohammed Mostafa
*/


#include "sc_fdma_demodulator.cuh"

__global__ void construct_fft_vec(hipfftComplex* pusch_bb_d, hipfftComplex* fft_vec_d, int M_pusch_sc) {

	int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int y_idx = blockIdx.y;
	
	if (y_idx == 0) //160 = N_cp_L_0
		fft_vec_d[x_idx] = pusch_bb_d[x_idx + 160]; 
	else if (y_idx == 7)// 14336 = y_idx * FFT_size   // 15520 =  y_idx * FFT_size + 2*N_cp_L_0 + 6*N_cp_L_else
		fft_vec_d[14336 + x_idx] = pusch_bb_d[x_idx + 15520]; 
	else if (y_idx < 7)
		fft_vec_d[y_idx * FFT_size + x_idx] = pusch_bb_d[x_idx + y_idx*FFT_size + N_cp_L_0 + y_idx*N_cp_L_else];
	else
		fft_vec_d[y_idx * FFT_size + x_idx] = pusch_bb_d[x_idx + y_idx*FFT_size + 176 + y_idx*N_cp_L_else]; // 176 = 2*N_cp_L_0 - N_cp_L_else
}

__global__ void extract_symbs(hipfftComplex* fft_vec_d, hipfftComplex* symbs_d, int M_pusch_sc_div2) {

	int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int y_idx = blockIdx.y;
	
	//if (x_idx >= M_pusch_sc)
		//return;
	symbs_d[y_idx*M_pusch_sc_div2*2 + x_idx] = fft_vec_d[y_idx*FFT_size + (x_idx + FFT_size - M_pusch_sc_div2)%FFT_size]; // 1448 = FFT_size - M_pusch_sc/2 
}

void sc_fdma_demodulator(hipfftComplex* pusch_bb_d, const int M_pusch_rb, hipfftComplex** symbs_d, hipfftHandle plan_sc_fdma, hipfftComplex* fft_vec_d)
{
	int M_pusch_sc = N_sc_rb * M_pusch_rb;
	
	//constructing fft_vec
	dim3 grid(2, N_symbs_per_subframe,1);
	dim3 block(1024,1,1);
	construct_fft_vec <<< grid, block >>>(pusch_bb_d, fft_vec_d, M_pusch_sc);

	// CUFFT plan
	hipfftExecC2C(plan_sc_fdma, fft_vec_d, fft_vec_d, HIPFFT_FORWARD);

	dim3 grid1(2, N_symbs_per_subframe, 1);
	dim3 block1(M_pusch_sc/2, 1, 1);
	extract_symbs << < grid1, block1 >> >(fft_vec_d, *symbs_d, M_pusch_sc/2);
	

	//Cleanup
	//hipFree(pusch_bb_d);
	//hipFree(fft_vec_d);
	//hipFree(symbs_d);

}
