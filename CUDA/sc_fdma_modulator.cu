#include "hip/hip_runtime.h"
/*
% Function:	sc_fdma_modulator
% Description:	Generates sc-fdma signal of the subframe
% Inputs:	*subframe_h:	received DMRS number 1
% 		M_pusch_rb	numer of resource blocks assigned to ue
% Outputs:	*pusch_bb_h	base band signal
By: Ahmad Nour & Mohammed Mostafa
*/


#include "sc_fdma_modulator.cuh"

__global__ void reshape_ifft_vec(hipfftComplex* subframe_d, hipfftComplex* ifft_vec_d, int M_pusch_sc) {

	int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int y_idx_ifft = blockIdx.y * FFT_size;
	int y_idx_subframe = blockIdx.y * M_pusch_sc;

	if (x_idx < 600) //M_pusch_sc / 2
		ifft_vec_d[y_idx_ifft + x_idx] = subframe_d[y_idx_subframe + x_idx + 600] / FFT_size;    // 600 = M_pusch_sc / 2
	else if (x_idx >= 1448) //FFT_size - M_pusch_sc / 2
		ifft_vec_d[y_idx_ifft + x_idx] = subframe_d[y_idx_subframe + x_idx - 1448] / FFT_size;   //1448 = FFT_size - M_pusch_sc / 2
	else
	{
		ifft_vec_d[y_idx_ifft + x_idx].x = 0;
		ifft_vec_d[y_idx_ifft + x_idx].y = 0;
	}
		
}

__global__ void add_cyclic_prefix(hipfftComplex* ifft_vec_d, hipfftComplex* pusch_bb_d, int M_pusch_sc) {

	int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int y_idx = blockIdx.y;

	//if (x_idx > 2207)  //2191 = FFT_size + N_cp_L_else - 1 = 2048 + 144 - 1
		//return;

	if (y_idx != 0 && y_idx != 7 && x_idx > 2191)  //2191 = FFT_size + N_cp_L_else - 1 = 2048 + 144 - 1
		return;

	if (y_idx == 0)
	{
		pusch_bb_d[x_idx] = ifft_vec_d[ (x_idx + FFT_size - N_cp_L_0)%FFT_size];
	}
	else if (y_idx == 7) // 15360 = FFT_size*7 + 144*6 + 160     // 14336 = y_idx*FFT_size = 7*2048
	{
		pusch_bb_d[15360 + x_idx] = ifft_vec_d[14336 + (x_idx + FFT_size - N_cp_L_0) % FFT_size];
	}
	else if (y_idx < 7)
	{
		pusch_bb_d[y_idx*FFT_size + N_cp_L_else*(y_idx-1)+ N_cp_L_0 + x_idx] = ifft_vec_d[y_idx*FFT_size + (x_idx + FFT_size - N_cp_L_else) % FFT_size];
	}
	else   //320 = 2*N_cp_L_0
	{
		pusch_bb_d[y_idx*FFT_size + N_cp_L_else*(y_idx - 2) + 320 + x_idx] = ifft_vec_d[y_idx*FFT_size + (x_idx + FFT_size - N_cp_L_else) % FFT_size];
	}
}

void sc_fdma_modulator(hipfftComplex* subframe_h, const int M_pusch_rb, hipfftComplex** pusch_bb_h)
{
	int M_pusch_sc = N_sc_rb * M_pusch_rb;
	
	//For timing purpose
	float elapsed = 0;				//For time calc.
	hipEvent_t start, stop;

	//Device data
	hipfftComplex* subframe_d;
	hipfftComplex* ifft_vec_d;
	hipfftComplex* pusch_bb_d;
	int* lookup_table_0_d;
	int* lookup_table_else_d;

	//Host data allocation
	*pusch_bb_h = (hipfftComplex *)malloc(sizeof(hipfftComplex)*modulated_subframe_length); 

	//Device data allocation
	startTimer();
	hipMalloc((void **)&subframe_d, sizeof(hipfftComplex)*N_symbs_per_subframe*M_pusch_sc);
	hipMalloc((void **)&ifft_vec_d, sizeof(hipfftComplex)*N_symbs_per_subframe*FFT_size);
	hipMalloc((void **)&pusch_bb_d, sizeof(hipfftComplex)*modulated_subframe_length);
	stopTimer("hipMalloc Time= %.6f ms\n", elapsed);

	//Copying data to device
	startTimer();
	hipMemcpy(subframe_d, subframe_h, sizeof(hipfftComplex)*N_symbs_per_subframe*M_pusch_sc, hipMemcpyHostToDevice);
	stopTimer("hipMemcpy Host->Device Time= %.6f ms\n", elapsed);

	//constructing fft_vec
	dim3 grid(2, N_symbs_per_subframe,1);
	dim3 block(1024,1,1);
	reshape_ifft_vec <<< grid, block >>>(subframe_d, ifft_vec_d, M_pusch_sc);

	// CUFFT plan
	int N_SIGS = N_symbs_per_subframe;
	int n[1] = { FFT_size };
	hipfftHandle plan;
	hipfftPlanMany(&plan, 1, n, NULL, 1, FFT_size, NULL, 1, FFT_size, HIPFFT_C2C, N_SIGS);
	hipfftExecC2C(plan, ifft_vec_d, ifft_vec_d, HIPFFT_BACKWARD);

	dim3 grid1(3, N_symbs_per_subframe, 1);
	dim3 block1(736, 1, 1);						//14*2048+14*160 = 736      //2048+160 = 736
	add_cyclic_prefix << < grid1, block1 >> >(ifft_vec_d, pusch_bb_d, M_pusch_sc);
	
	
	//Retrieve data from device
	startTimer();
	hipMemcpy(*pusch_bb_h, pusch_bb_d, sizeof(hipfftComplex)*modulated_subframe_length, hipMemcpyDeviceToHost);
	stopTimer("hipMemcpy Device->Host Time= %.6f ms\n", elapsed);

	// Cleanup
	hipFree(subframe_d);
	hipFree(ifft_vec_d);

	//Destroy timers
	destroyTimers();
}
