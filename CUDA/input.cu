/*
This function is used to read the bits from a text file and return the number of bits as well.
If no argument is passed, it will try to read from the same directory from the file "input.txt".
Bits should be in one line with no spaces, for example: (100011110)
To do so in MATLAB, use this command:

dlmwrite('output.txt',Variable_to_Print,'delimiter','');

By: Ahmad Nour
*/

#include "input.cuh"

Byte* readBits(int argc, char* argv, int *numBits)
{

	FILE *inputFile;
	char* path = "input.txt";

	if (argc >= 2)		//a path is given, use it instead
		path = argv;

	if ((inputFile = fopen(path, "r+")) == NULL) {
		printf("Cannot open file.\n");
		exit(1);
	}

	fseek(inputFile, 0, SEEK_END);
	long N = ftell(inputFile);
	fseek(inputFile, 0, SEEK_SET);

	Byte* inputBits = (Byte*)malloc(sizeof(Byte)* N);

	fread(inputBits, sizeof(char), N, inputFile);

	fclose(inputFile);

	*numBits = N;

	for (int i = 0; i < N; i++)
	{
		inputBits[i] -= '0';
	}

	return inputBits;

}
