#include "hip/hip_runtime.h"
/*
% Function:		transform_precoder
% Description:	perform transform precoding on complex data after mapper
% Inputs:		*symbols_R_h:	Real part of the symbols
% Inputs:		*symbols_I_h:	Imag part of the symbols
%				M_pusch_rb		numer of resource blocks assigned to ue
% Outputs:		*precoded_data	transform precodded data
By: Ahmad Nour & Mohammed Mostafa
*/

/*
coeff_multiply kernel just multiples the output symbols by a coeff. The kernel's overhead can be avoided if we
merged it with the mapper kernel
*/

#include "transform_precoder.cuh"

__global__ void coeff_multiply(hipfftComplex* symbols_d, double coeff, int numThreads) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//Not to run more threads than available data
	if (idx >= numThreads)
		return;

	symbols_d[idx].x *= coeff;
	symbols_d[idx].y *= coeff;
}

void transform_precoder(hipfftComplex* symbols_h, const int M_pusch_rb, int signal_size, hipfftComplex** precoded_data_h)
{
	int M_pusch_sc = N_sc_rb * M_pusch_rb;

	//For timing purpose
	float elapsed = 0;				//For time calc.
	hipEvent_t start, stop;

	//Device data
	hipfftComplex* symbols_d;
	hipfftComplex* precoded_data_d;

	//Host data allocation
	*precoded_data_h = (hipfftComplex *)malloc(sizeof(hipfftComplex)*signal_size);
	
	//Device data allocation
	startTimer();
	hipMalloc((void **)&symbols_d, sizeof(hipfftComplex)*signal_size);
	hipMalloc((void **)&precoded_data_d, sizeof(hipfftComplex)*signal_size);
	stopTimer("hipMalloc Time= %.6f ms\n", elapsed);

	//Copying data to device
	startTimer();
	hipMemcpy(symbols_d, symbols_h, sizeof(hipfftComplex)*signal_size, hipMemcpyHostToDevice);
	stopTimer("hipMemcpy Host->Device Time= %.6f ms\n", elapsed);

	// CUFFT plan
	int N_SIGS = signal_size / M_pusch_sc;
	int n[1] = { M_pusch_sc };
	hipfftHandle plan;

	hipfftPlanMany(&plan, 1, n, NULL, 1, M_pusch_sc, NULL, 1, M_pusch_sc, HIPFFT_C2C, N_SIGS);
	hipfftExecC2C(plan, symbols_d, precoded_data_d, HIPFFT_FORWARD);

	//Calc. number of needed threads for calling kernel(s)
	int numThreads = signal_size;
	int blockDim = (numThreads < 1024) ? numThreads : 1024;	//block size in threads (max 1024 thread)
	int gridDim = numThreads / (blockDim)+(numThreads % blockDim == 0 ? 0 : 1); //grid size in bloack (min 1)

	//Coeff. Multiplication
	coeff_multiply << <gridDim, blockDim >> > (precoded_data_d, rsqrtf(M_pusch_sc), numThreads);

	//Retrieve data from device
	startTimer();
	hipMemcpy(*precoded_data_h, precoded_data_d, sizeof(hipfftComplex)*signal_size, hipMemcpyDeviceToHost);
	stopTimer("hipMemcpy Device->Host Time= %.6f ms\n", elapsed);

	// Cleanup
	hipFree(symbols_d);
	hipFree(precoded_data_d);

	//Destroy timers
	destroyTimers();
}
