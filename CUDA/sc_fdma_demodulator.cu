#include "hip/hip_runtime.h"
/*
% Function:		sc_fdma_demodulator
% Description:	Generates complex symbols from the sc-fdma symbols
% Inputs		*pusch_bb_h		sc-fdma symbols
M_pusch_rb		numer of resource blocks assigned to ue
% Outputs:		*symbs_h		output symbols
By: Ahmad Nour & Mohammed Mostafa
*/
/*
coeff_multiply kernel just multiples the output symbols by a coeff. The kernel's overhead can be avoided if we
merged it with the mapper kernel
*/

#include "sc_fdma_demodulator.cuh"

__global__ void construct_fft_vec(hipfftComplex* pusch_bb_d, hipfftComplex* fft_vec_d, int M_pusch_sc) {

	int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int y_idx = blockIdx.y;
	
	if (y_idx == 0) //160 = N_cp_L_0
		fft_vec_d[x_idx] = pusch_bb_d[x_idx + 160]; 
	else if (y_idx == 7)// 14336 = y_idx * FFT_size   // 15520 =  y_idx * FFT_size + 2*N_cp_L_0 + 6*N_cp_L_else
		fft_vec_d[14336 + x_idx] = pusch_bb_d[x_idx + 15520]; 
	else if (y_idx < 7)
		fft_vec_d[y_idx * FFT_size + x_idx] = pusch_bb_d[x_idx + y_idx*FFT_size + N_cp_L_0 + y_idx*N_cp_L_else];
	else
		fft_vec_d[y_idx * FFT_size + x_idx] = pusch_bb_d[x_idx + y_idx*FFT_size + 176 + y_idx*N_cp_L_else]; // 176 = 2*N_cp_L_0 - N_cp_L_else
}

__global__ void extract_symbs(hipfftComplex* fft_vec_d, hipfftComplex* symbs_d, int M_pusch_sc_div2) {

	int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int y_idx = blockIdx.y;
	
	//if (x_idx >= M_pusch_sc)
		//return;
	symbs_d[y_idx*M_pusch_sc_div2*2 + x_idx] = fft_vec_d[y_idx*FFT_size + (x_idx + FFT_size - M_pusch_sc_div2)%FFT_size]; // 1448 = FFT_size - M_pusch_sc/2 
}

void sc_fdma_demodulator(hipfftComplex* pusch_bb_h, const int M_pusch_rb, hipfftComplex** symbs_h)
{
	int M_pusch_sc = N_sc_rb * M_pusch_rb;
	
	//For timing purpose
	float elapsed = 0;				//For time calc.
	hipEvent_t start, stop;

	//Device data
	hipfftComplex* pusch_bb_d;
	hipfftComplex* fft_vec_d;
	hipfftComplex* symbs_d;

	//Host data allocation
	*symbs_h = (hipfftComplex *)malloc(sizeof(hipfftComplex)*N_symbs_per_subframe*M_pusch_sc);

	//Device data allocation
	startTimer();
	hipMalloc((void **)&pusch_bb_d, sizeof(hipfftComplex)*modulated_subframe_length);
	hipMalloc((void **)&fft_vec_d, sizeof(hipfftComplex)*N_symbs_per_subframe*FFT_size);
	hipMalloc((void **)&symbs_d, sizeof(hipfftComplex)*N_symbs_per_subframe*M_pusch_sc);
	stopTimer("hipMalloc Time= %.6f ms\n", elapsed);

	//Copying data to device
	startTimer();
	hipMemcpy(pusch_bb_d, pusch_bb_h, sizeof(hipfftComplex)*modulated_subframe_length, hipMemcpyHostToDevice);
	stopTimer("hipMemcpy Host->Device Time= %.6f ms\n", elapsed);

	//constructing fft_vec
	dim3 grid(2, N_symbs_per_subframe,1);
	dim3 block(1024,1,1);
	construct_fft_vec <<< grid, block >>>(pusch_bb_d, fft_vec_d, M_pusch_sc);

	// CUFFT plan
	int N_SIGS = N_symbs_per_subframe;
	int n[1] = { FFT_size };
	hipfftHandle plan;
	hipfftPlanMany(&plan, 1, n, NULL, 1, FFT_size, NULL, 1, FFT_size, HIPFFT_C2C, N_SIGS);
	hipfftExecC2C(plan, fft_vec_d, fft_vec_d, HIPFFT_FORWARD);

	dim3 grid1(2, N_symbs_per_subframe, 1);
	dim3 block1(M_pusch_sc/2, 1, 1);
	extract_symbs << < grid1, block1 >> >(fft_vec_d, symbs_d, M_pusch_sc/2);
	
	
	//Retrieve data from device
	startTimer();
	hipMemcpy(*symbs_h, symbs_d, sizeof(hipfftComplex)*N_symbs_per_subframe*M_pusch_sc, hipMemcpyDeviceToHost);
	stopTimer("hipMemcpy Device->Host Time= %.6f ms\n", elapsed);

	// Cleanup
	hipFree(pusch_bb_d);
	hipFree(fft_vec_d);
	hipFree(symbs_d);
	//Destroy timers
	destroyTimers();
}
