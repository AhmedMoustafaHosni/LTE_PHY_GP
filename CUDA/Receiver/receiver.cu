#include "hip/hip_runtime.h"
/*
% Function:    receiver
% Inputs:      modulated_subframe       - modulated subframe (modulated symbols)
%              M_pusch_rb               - number of resource blocks assigned to the UE
% Outputs:     received_bits			- received bits
By: Mohammed Mostafa
*/

#include "sc_fdma_demodulator.cuh"
#include "generate_dmrs_pusch.cuh"
#include "generate_ul_rs.cuh"
#include "generate_psuedo_random_seq.cuh"
#include "transform_predecoder.cuh"
#include "decompose_subframe.cuh"
#include "demapper.cuh"
#include "descrambler.cuh"
#include "deinterleaver.cuh"

int main(int argc, char **argv) {

	const int Qm = 2;				// Modulation Order(2 = QPSK, 4 = 16QAM, 6 = 64QAM)
	const int M_pusch_rb = 100;		//number of resource blocks assigned to the UE
	const int N_l = 1;				// Number of Layers
	const int N_ri = 12;			//length of ri symbols
	const int n_s = 0;				//assume UE send on subframe 0
	const int N_id_cell = 2;		//assume enodeB scheduled cell 2 for the UE
	const int M_pusch_sc = N_sc_rb * M_pusch_rb; //total number of subcarriers
	const int n_RNTI = 10;			//radio network temporary identifier given to the UE by enodeB (assume 10)
	const int N_bits = Qm * 12 * M_pusch_sc;   //Qm * 12 * M_pusch_sc = 2*12*1200

	//input
	hipfftComplex* subframe_h = (hipfftComplex *)malloc(sizeof(hipfftComplex)*modulated_subframe_length);

	for (int i = 0; i < modulated_subframe_length; i++)
	{
		subframe_h[i].x = rand() / (float)RAND_MAX * 100;
		subframe_h[i].y = rand() / (float)RAND_MAX * 100;
	}


	hipfftComplex* subframe_d;
	hipMalloc((void **)&subframe_d, sizeof(hipfftComplex)*modulated_subframe_length);
	hipMemcpy(subframe_d, subframe_h, sizeof(hipfftComplex)*modulated_subframe_length, hipMemcpyHostToDevice);

	//Device data allocation
	hipfftComplex* fft_vec_d;
	hipfftComplex* demod_subframe_d;
	hipMalloc((void **)&fft_vec_d, sizeof(hipfftComplex)*N_symbs_per_subframe*FFT_size);
	hipMalloc((void **)&demod_subframe_d, sizeof(hipfftComplex)*N_symbs_per_subframe*M_pusch_sc);

	hipfftComplex* x_q_d;
	hipfftComplex* dmrs1_generated_d = 0, *dmrs2_generated_d = 0;
	hipMalloc((void **)&dmrs1_generated_d, sizeof(hipfftComplex)*N_sc_rb*M_pusch_rb);
	hipMalloc((void **)&dmrs2_generated_d, sizeof(hipfftComplex)*N_sc_rb*M_pusch_rb);
	hipMalloc((void **)&x_q_d, sizeof(hipfftComplex)*prime_nums[M_pusch_rb - 1]);

	hipfftComplex* dmrs1_decomposed_d;
	hipfftComplex* dmrs2_decomposed_d;
	hipfftComplex* complex_data_d;
	hipMalloc((void **)&complex_data_d, sizeof(hipfftComplex) * 12 * M_pusch_sc);
	hipMalloc((void **)&dmrs1_decomposed_d, sizeof(hipfftComplex)*M_pusch_sc);
	hipMalloc((void **)&dmrs2_decomposed_d, sizeof(hipfftComplex)*M_pusch_sc);

	hipfftComplex* predecoded_data_d;
	hipMalloc((void **)&predecoded_data_d, sizeof(hipfftComplex)* 12 * M_pusch_sc);

	Byte *bits_d;
	hipMalloc((void **)&bits_d, sizeof(Byte)* Qm * 12 * M_pusch_sc);    //FIX Number_demaped_bits

	Byte* c_d = 0;
	hipMalloc((void **)&c_d, sizeof(Byte)*Qm * 12 * M_pusch_sc);
	Byte *descrambled_bits_d;
	hipMalloc((void **)&descrambled_bits_d, sizeof(Byte)* Qm * 12 * M_pusch_sc);


	// Step 1: Define C_mux
	int C_mux = N_pusch_symbs;
	// Step 2: Define R_mux and R_prime_mux
	int H_prime_total = N_bits / (Qm*N_l);
	int H_prime = H_prime_total - N_ri;
	int R_mux = (H_prime_total*Qm*N_l) / C_mux;
	int R_prime_mux = R_mux / (Qm*N_l);

	Byte *ri_d, *y_idx_d, *y_mat_d;
	Byte *received_bits_d;
	hipMalloc((void **)&ri_d, sizeof(Byte)*(N_ri * Qm * N_l));
	hipMalloc((void **)&y_idx_d, sizeof(Byte)*(C_mux*R_prime_mux));
	hipMalloc((void **)&y_mat_d, sizeof(Byte)*(C_mux*R_mux));
	hipMalloc((void **)&received_bits_d, sizeof(Byte)* H_prime * Qm * N_l);

	//create plans
	int n[1] = { FFT_size };
	hipfftHandle plan_sc_fdma;
	hipfftPlanMany(&plan_sc_fdma, 1, n, NULL, 1, FFT_size, NULL, 1, FFT_size, HIPFFT_C2C, N_symbs_per_subframe);

	int N_SIGS = 12;   //signal_size/M_pusch_sc = 12 * M_pusch_sc / M_pusch_sc = 12
	n[0] = { M_pusch_sc };
	hipfftHandle plan_transform_predecoder;
	hipfftPlanMany(&plan_transform_predecoder, 1, n, NULL, 1, M_pusch_sc, NULL, 1, M_pusch_sc, HIPFFT_C2C, N_SIGS);
	
	//sc-fdma demodulation
	sc_fdma_demodulator(subframe_d, M_pusch_rb, &demod_subframe_d, plan_sc_fdma, fft_vec_d);

	//generate dmrs   
	generate_dmrs_pusch(n_s, N_id_cell, 0, 0, 0, 0, 0, "fixed", M_pusch_rb, 0, &dmrs1_generated_d, &dmrs2_generated_d, x_q_d);

	//Decompose subframe
	decompose_subframe(demod_subframe_d, M_pusch_rb, &complex_data_d, &dmrs1_decomposed_d, &dmrs2_decomposed_d);

	//FIX  "CHANNEL ESTIMATION" SECTIONS
	

	//predecoding   
	transform_predecoder(complex_data_d, M_pusch_rb, 12 * M_pusch_sc , &predecoded_data_d, plan_transform_predecoder); //signal_size = 12 * M_pusch_sc

	//demapping
	demapper(predecoded_data_d, &bits_d, Qm * 12 * M_pusch_sc , Qm);  //Number_demaped_bits = Qm * 12 * M_pusch_sc

	
	//Generate Pseudo Random Seq.
	Byte *c_h = 0;
	generate_psuedo_random_seq(&c_h, N_bits, n_RNTI, n_s, N_id_cell);
	//Copy (c) to Device
	hipMemcpy(c_d, c_h, sizeof(Byte)*N_bits, hipMemcpyHostToDevice);
	//Descrammpling
	descrambler(bits_d, &descrambled_bits_d, c_d, N_bits);

	//deinterleaver
	deinterleaver(descrambled_bits_d, &ri_d, &received_bits_d, N_bits, N_ri, Qm, N_l, y_idx_d, y_mat_d);

	//Retrieve data from device
	Byte* received_bits_h = (Byte*)malloc(sizeof(Byte)*H_prime * Qm * N_l);
	hipMemcpy(received_bits_h, received_bits_d, sizeof(Byte)*H_prime * Qm * N_l, hipMemcpyDeviceToHost);
	Byte* ri_h = (Byte*)malloc(sizeof(Byte)*N_ri * Qm * N_l);
	hipMemcpy(ri_h, ri_d, sizeof(Byte)*N_ri * Qm * N_l, hipMemcpyDeviceToHost);

	//Print results
	for (int i = 0; i < H_prime * Qm * N_l; i++)
	{
			printf("idx = %d \t %d  \n", i + 1, received_bits_h[i]);
	}
	

	//test file
	FILE *results1;
	if ((results1 = freopen("Receiver_test.m", "w+", stdout)) == NULL) {
		printf("Cannot open file.\n");
		exit(1);
	}

	//input subframe
	printf("clear; clc;\nsymbols_in_real = [ ");
	for (int i = 0; i < (modulated_subframe_length); i++)
	{
		printf("%10f", subframe_h[i].x);
		if (i != ((modulated_subframe_length)-1))
			printf(",");
	}
	printf(" ];\nsymbols_in_imag = [ ");
	for (int i = 0; i < (modulated_subframe_length); i++)
	{
		printf("%10f", subframe_h[i].y);
		if (i != ((modulated_subframe_length)-1))
			printf(",");
	}
	printf(" ];\n");
	printf("subframe_input_CUDA = symbols_in_real + 1i * symbols_in_imag;\n");

	//Received Bits
	printf("\nReceved_bits_cuda = [ ");
	for (int i = 0; i < (H_prime * Qm * N_l); i++)
	{
		printf("%d", received_bits_h[i]);
		if (i != ((Qm * 12 * M_pusch_sc)-1))
			printf(",");
	}
	printf(" ];\n");

	//RI Bits
	printf("\nRI_bits_cuda = [ ");
	for (int i = 0; i < (N_ri * Qm * N_l); i++)
	{
		printf("%d", ri_h[i]);
		if (i != ((N_ri * Qm * N_l) - 1))
			printf(",");
	}
	printf(" ];\n");

	printf("N_id_cell = 2;N_sc_rb   = 12;M_pusch_rb = 100;M_pusch_sc = M_pusch_rb*N_sc_rb;Nc = 1600;n_s = 0;n_RNTI = 10;M_bits = 2*1200*12;N_l = 1;N_ri_bits = 12;N_ack_bits =0;Q_m = 2;modulated_subframe = subframe_input_CUDA;demodulated_subframe = sc_fdma_demodulator(modulated_subframe, M_pusch_rb);demodulated_subframe_vect =[demodulated_subframe(0+1,:), demodulated_subframe(1+1,:), demodulated_subframe(2+1,:), demodulated_subframe(4+1,:), demodulated_subframe(5+1,:), demodulated_subframe(6+1,:), demodulated_subframe(7+1,:), demodulated_subframe(8+1,:), demodulated_subframe(9+1,:), demodulated_subframe(11+1,:), demodulated_subframe(12+1,:), demodulated_subframe(13+1,:)];dmrs = generate_dmrs_pusch(n_s, N_id_cell, 0, 0, 0, 0, 0, 'fixed', M_pusch_rb, 0);dmrs_1 = dmrs(1:M_pusch_sc);dmrs_2 = dmrs(M_pusch_sc+1:2*M_pusch_sc);dmrs_1_rx = demodulated_subframe(1+3,:);dmrs_2_rx = demodulated_subframe(1+10,:);predecoded_data = transform_predecoder(demodulated_subframe_vect, M_pusch_rb);demapped_data = demapper_hard(predecoded_data, 'qpsk' );c_init = n_RNTI * 2^14 + floor(n_s/2) * 2^9 + N_id_cell;c = generate_psuedo_random_seq(c_init, M_bits);descrambled_bits = descramble(demapped_data, c);[data_bits, ri_bits, ack_bits] = channel_deinterleaver(descrambled_bits, N_ri_bits, N_ack_bits, N_l, Q_m);isequal(data_bits,Receved_bits_cuda)\nisequal(ri_bits, RI_bits_cuda)");

	fclose(results1);
	return 0;
}