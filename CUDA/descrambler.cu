#include "hip/hip_runtime.h"
/*
% Function:		descrambler
% Description:	descramble bits with psuedo random seq.
% Inputs:		bits_h:				Binary bits to descramble
%				c_h:				psuedo random sequence
% Outputs:		*descrambledbits_h:	Descrambled Bits
By: Ahmad Nour & Mohammed Mostafa
*/

#include "descrambler.cuh"

__global__ void descrabmler(Byte *bits_d, Byte *descrambledbits_d, Byte *c_d, int numThreads)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//Not to run more threads than available data
	if (idx >= numThreads)
		return;

	descrambledbits_d[idx] = bits_d[idx] ^ c_d[idx];
}

void descrambler(Byte* bits_h, Byte** descrambledbits_h, const Byte* c_h, const int N)
{
	//For timing purpose
	float elapsed = 0;				//For time calc.
	hipEvent_t start, stop;

	//Device data
	Byte *bits_d;
	Byte *descrambledbits_d;
	Byte *c_d;

	//Host data allocation
	*descrambledbits_h = (Byte *)malloc(sizeof(Byte)*N);

	//Device data allocation
	startTimer();
	hipMalloc((void **)&bits_d, sizeof(Byte)*N);
	hipMalloc((void **)&descrambledbits_d, sizeof(Byte)*N);
	hipMalloc((void **)&c_d, sizeof(Byte)*N);
	stopTimer("hipMalloc Time= %.6f ms\n", elapsed);

	//Copying data to device
	startTimer();
	hipMemcpy(bits_d, bits_h, sizeof(Byte)*N, hipMemcpyHostToDevice);
	hipMemcpy(c_d, c_h, sizeof(Byte)*N, hipMemcpyHostToDevice);
	stopTimer("hipMemcpy Host->Device Time= %.6f ms\n", elapsed);

	//Calc. number of needed threads for calling kernel(s)
	int numThreads = N;
	int blockDim = (numThreads < 1024) ? numThreads : 1024;	//block size in threads (max 1024 thread)
	int gridDim = numThreads / (blockDim)+(numThreads % blockDim == 0 ? 0 : 1); //grid size in bloack (min 1)

	//Calling the kernel(s)
	startTimer();
	descrabmler << <gridDim, blockDim >> > (bits_d, descrambledbits_d, c_d, N);
	stopTimer("Scrambler Time= %.6f ms\n", elapsed);

	//Retrieve data from device
	startTimer();
	hipMemcpy(*descrambledbits_h, descrambledbits_d, sizeof(Byte)*N, hipMemcpyDeviceToHost);
	stopTimer("hipMemcpy Device->Host Time= %.6f ms\n", elapsed);

	// Cleanup
	hipFree(bits_d);
	hipFree(descrambledbits_d);
	hipFree(c_d);

	//Destroy timers
	destroyTimers();
}
