#include "hip/hip_runtime.h"
/*
% Function:		compose_subframe
% Description:	compose the subframe by multiplexing the dmrs signal and data
% Inputs:		*complex_data_h:	complex data to be sent in subframe
%				*dmrs_1_h:			demodulation reference signal number 1
%				*dmrs_2_h:			demodulation reference signal number 2
%				M_pusch_rb			number of resource blocks assigned to the ue
% Outputs:		*subframe_h			the subframe with data of all ofdm symbols
By: Ahmad Nour & Mohammed Mostafa
*/

/*
coeff_multiply kernel just multiples the output symbols by a coeff. The kernel's overhead can be avoided if we
merged it with the mapper kernel
*/

#include "compose_subframe.cuh"

__global__ void compose_subframe(hipfftComplex* complex_data_d, hipfftComplex* dmrs_1_d, hipfftComplex* dmrs_2_d, int M_pusch_sc, hipfftComplex* subframe_d) {

	int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int y_idx = blockIdx.y;

	if (y_idx == 3)
		subframe_d[y_idx*M_pusch_sc + x_idx] = dmrs_1_d[x_idx];
	else if (y_idx == 10)
		subframe_d[y_idx*M_pusch_sc + x_idx] = dmrs_2_d[x_idx];
	else if (y_idx > 10)
		subframe_d[y_idx*M_pusch_sc + x_idx] = complex_data_d[(y_idx - 2)*M_pusch_sc + x_idx];
	else if (y_idx > 3)
		subframe_d[y_idx*M_pusch_sc + x_idx] = complex_data_d[(y_idx - 1)*M_pusch_sc + x_idx];
	else
		subframe_d[y_idx*M_pusch_sc + x_idx] = complex_data_d[y_idx*M_pusch_sc + x_idx];

}



void compose_subframe(hipfftComplex* complex_data_h, hipfftComplex* dmrs_1_h, hipfftComplex* dmrs_2_h, const int M_pusch_rb, hipfftComplex** subframe_h)
{
	int M_pusch_sc = N_sc_rb * M_pusch_rb;

	//For timing purpose
	float elapsed = 0;				//For time calc.
	hipEvent_t start, stop;

	//Device data
	hipfftComplex* complex_data_d;
	hipfftComplex* dmrs_1_d;
	hipfftComplex* dmrs_2_d;
	hipfftComplex* subframe_d;

	//Host data allocation
	*subframe_h = (hipfftComplex *)malloc(sizeof(hipfftComplex)*N_symbs_per_subframe*M_pusch_sc);

	//Device data allocation
	startTimer();
	hipMalloc((void **)&complex_data_d, sizeof(hipfftComplex) * 12 * M_pusch_sc);
	hipMalloc((void **)&dmrs_1_d, sizeof(hipfftComplex)*M_pusch_sc);
	hipMalloc((void **)&dmrs_2_d, sizeof(hipfftComplex)*M_pusch_sc);
	hipMalloc((void **)&subframe_d, sizeof(hipfftComplex)*N_symbs_per_subframe*M_pusch_sc);
	stopTimer("hipMalloc Time= %.6f ms\n", elapsed);

	//Copying data to device
	startTimer();
	hipMemcpy(complex_data_d, complex_data_h, sizeof(hipfftComplex) * 12 * M_pusch_sc, hipMemcpyHostToDevice);
	hipMemcpy(dmrs_1_d, dmrs_1_h, sizeof(hipfftComplex) * M_pusch_sc, hipMemcpyHostToDevice);
	hipMemcpy(dmrs_2_d, dmrs_2_h, sizeof(hipfftComplex) * M_pusch_sc, hipMemcpyHostToDevice);
	stopTimer("hipMemcpy Host->Device Time= %.6f ms\n", elapsed);

	//compose subframe
	dim3 grid(2, N_symbs_per_subframe, 1);
	dim3 block(M_pusch_sc / 2, 1, 1);
	compose_subframe << < grid, block >> >(complex_data_d, dmrs_1_d, dmrs_2_d, M_pusch_sc, subframe_d);

	//Retrieve data from device
	startTimer();
	hipMemcpy(*subframe_h, subframe_d, sizeof(hipfftComplex)*N_symbs_per_subframe*M_pusch_sc, hipMemcpyDeviceToHost);
	stopTimer("hipMemcpy Device->Host Time= %.6f ms\n", elapsed);

	// Cleanup
	hipFree(complex_data_d);
	hipFree(dmrs_1_d);
	hipFree(dmrs_2_d);
	hipFree(subframe_d);

	//Destroy timers
	destroyTimers();
}